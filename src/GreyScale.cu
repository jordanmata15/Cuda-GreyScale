#include "hip/hip_runtime.h"
#include "GreyScale.cuh"
#include <iostream>
#include <cstring>

#include <cstdio>
#include <jpeglib.h>
#include <jerror.h>

using namespace cimg_library;


GreyScale::GreyScale(){
    //std::filesystem::path dir ("..");
    //this->filePath = dir / "data";
    this->filePath = "../data";
    this->fileName = "dino.bmp";
}


GreyScale::GreyScale(std::string filePath, std::string fileName){
    this->filePath = filePath;
    this->fileName = fileName;
}


void GreyScale::loadFile(){
    //std::string fullFilePath = this->filePath / this->fileName;
    std::string fullFilePath = this->filePath + "/" + this->fileName;
    const char* filename_input = fullFilePath.c_str();
    this->img = CImg<u_char>(filename_input);
}


void GreyScale::makeGreyScaleSerial(){
    int width  = this->img.width(),
        height = this->img.height(), 
        depth  = this->img.depth();

    /* pointer to image pixels. Colors are rranged in contiguous memory. 
    Eg: {RRR...RRRGGG...GGGBBB...BBB} */
    auto arr = this->img.data(); 
    size_t length = width*height*depth;
    size_t rgbOffset = length;
    // pointers to the start of each of the color channels
    u_char* R = &arr[0*rgbOffset];
    u_char* G = &arr[1*rgbOffset];
    u_char* B = &arr[2*rgbOffset];

    for (size_t i=0; i<length; ++i){
        u_char greyScaleValue = R[i]*0.3 + G[i]*0.59 + B[i]*0.11;
        R[i] = greyScaleValue;
        G[i] = greyScaleValue;
        B[i] = greyScaleValue;
    }
}


void GreyScale::makeGreyScaleParallel(){
    /* pointer to image pixels. Colors are rranged in contiguous memory. 
    Eg: {RRR...RRRGGG...GGGBBB...BBB} */
    u_char *hostImageArr = this->img.data();
    u_char *devImageArr;
    int *devRGBOffset;

    size_t  channels = 3,
            length = this->img.width() * this->img.height() * this->img.depth() * channels,
            rgbOffset = length/channels;

    hipMalloc((void**)&devImageArr, length*sizeof(u_char));
    hipMalloc((void**)&devRGBOffset, sizeof(int));
    hipMemcpy(devImageArr, hostImageArr, length*sizeof(u_char), hipMemcpyHostToDevice);
    hipMemcpy(devRGBOffset, &rgbOffset, sizeof(int), hipMemcpyHostToDevice);

    mykernel<<<this->blockSize,this->gridSize>>>(devImageArr, devRGBOffset);

    hipMemcpy(hostImageArr, devImageArr, length*sizeof(u_char), hipMemcpyDeviceToHost);
    hipFree(devImageArr);
    hipFree(devRGBOffset);
}


__global__ void mykernel(u_char* imageArr, int* devRGBOffset){
    int rgbOffset = *devRGBOffset;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    u_char  *R = &imageArr[0*rgbOffset],
            *G = &imageArr[1*rgbOffset],
            *B = &imageArr[2*rgbOffset],
            greyScaleValue;

    if (index < rgbOffset){
        greyScaleValue = R[index]*0.3 + G[index]*0.59 + B[index]*0.11;
        R[index] = greyScaleValue;
        G[index] = greyScaleValue;
        B[index] = greyScaleValue;
    }
}


void GreyScale::display(){
    this->img.display();
}


void GreyScale::writeFile(){
    std::string outputFileName = "GreyScale_" + fileName;
    //std::filesystem::path outFilePath = this->filePath / outputFileName;
    std::string outFilePath = this->filePath + "/" + outputFileName;
    const char * filename_output = outFilePath.c_str();
    this->img.save(filename_output);
}